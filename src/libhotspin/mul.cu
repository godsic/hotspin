#include "hip/hip_runtime.h"
#include "mul.h"

#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void mulKern(float* dst, float* a, float* b, int Npart)
{
    int i = threadindex;
    if (i < Npart)
    {
        dst[i] = a[i] * b[i];
    }
}


__export__ void mulAsync(float* dst, float* a, float* b, hipStream_t stream, int Npart)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    mulKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (dst, a, b, Npart);
}

#ifdef __cplusplus
}
#endif

