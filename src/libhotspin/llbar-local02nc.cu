#include "hip/hip_runtime.h"
#include "llbar-local02nc.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif

    __global__ void llbarLocal02ncKern(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                                       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                                       float* __restrict__ hx, float* __restrict__ hy, float* __restrict__ hz,

                                       float* __restrict__ msat0T0Msk,

                                       float* __restrict__ mu_xx,
                                       float* __restrict__ mu_yy,
                                       float* __restrict__ mu_zz,

                                       const float muMul_xx,
                                       const float muMul_yy,
                                       const float muMul_zz,

                                       int Npart)
    {

        int x0 = threadindex;

        if (x0 < Npart)
        {

            float msat0T0 = (msat0T0Msk == NULL) ? 1.0 : msat0T0Msk[x0];
            float3 m = make_float3(mx[x0], my[x0], mz[x0]);

            // make sure there is no torque for non-magnetic points
            if (msat0T0 == 0.0f)
            {
                tx[x0] = 0.0f;
                ty[x0] = 0.0f;
                tz[x0] = 0.0f;
                return;
            }

            float3 H = make_float3(hx[x0], hy[x0], hz[x0]);

            float _mH = dotf(m, H);

            float3 _mHm;

            _mHm.x = _mH * m.x;
            _mHm.y = _mH * m.y;
            _mHm.z = _mH * m.z;

            float3 _mu_mHm;

            float m_xx = (mu_xx != NULL) ? mu_xx[x0] * muMul_xx : muMul_xx;

            _mu_mHm.x = m_xx * _mHm.x;

            float m_yy = (mu_yy != NULL) ? mu_yy[x0] * muMul_yy : muMul_yy;

            _mu_mHm.y = m_yy * _mHm.y;

            float m_zz = (mu_zz != NULL) ? mu_zz[x0] * muMul_zz : muMul_zz;

            _mu_mHm.z = m_zz * _mHm.z;

            tx[x0] = _mu_mHm.x;
            ty[x0] = _mu_mHm.y;
            tz[x0] = _mu_mHm.z;
        }
    }

    __export__  void llbar_local02nc_async(float* tx, float*  ty, float*  tz,
                                           float*  mx, float*  my, float*  mz,
                                           float*  hx, float*  hy, float*  hz,

                                           float* msat0T0,

                                           float* mu_xx,
                                           float* mu_yy,
                                           float* mu_zz,

                                           const float muMul_xx,
                                           const float muMul_yy,
                                           const float muMul_zz,

                                           hipStream_t stream,
                                           int Npart)
    {
        dim3 gridSize, blockSize;
        make1dconf(Npart, &gridSize, &blockSize);
        llbarLocal02ncKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (tx, ty, tz,
                mx, my, mz,
                hx, hy, hz,

                msat0T0,

                mu_xx,
                mu_yy,
                mu_zz,

                muMul_xx,
                muMul_yy,
                muMul_zz,

                Npart);

    }

// ========================================

#ifdef __cplusplus
}
#endif
