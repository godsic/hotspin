#include "hip/hip_runtime.h"

#include "temperature.h"

#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void temperature_scaleAnisKern(float* __restrict__ hx, float* __restrict__ hy, float* __restrict__ hz,

        float* __restrict__ mu_xx,
        float* __restrict__ mu_yy,
        float* __restrict__ mu_zz,

        float* __restrict__ tempMask,
        float* __restrict__ msat0T0Mask,

        const float muMul_xx,
        const float muMul_yy,
        const float muMul_zz,

        const float KB2tempMul_mu0VgammaDtMSatMul,

        int Npart)
{


    int i = threadindex;

    if (i < Npart)
    {

        float msat0T0 = getMaskUnity(msat0T0Mask, i);
        if (msat0T0 == 0.0f)
        {
            hx[i] = 0.0f;
            hy[i] = 0.0f;
            hz[i] = 0.0f;
            return;
        }

        float3 H = make_float3(hx[i], hy[i], hz[i]);

        float3 mu_H;

        float m_xx = muMul_xx * getMaskUnity(mu_xx, i);
        m_xx = sqrtf(m_xx);
        mu_H.x = m_xx * H.x;

        float m_yy = muMul_yy * getMaskUnity(mu_yy, i);
        m_yy = sqrtf(m_yy);
        mu_H.y = m_yy * H.y;

        float m_zz = muMul_zz * getMaskUnity(mu_zz, i);
        m_zz = sqrtf(m_zz);
        mu_H.z = m_zz * H.z;

        float T = getMaskUnity(tempMask, i);
        float pre = sqrtf((T * KB2tempMul_mu0VgammaDtMSatMul) / msat0T0);
        
        hx[i] = pre * mu_H.x;
        hy[i] = pre * mu_H.y;
        hz[i] = pre * mu_H.z;

    }
}


__export__ void temperature_scaleAnizNoise(float* hx, float* hy, float* hz,
        float* mu_xx,
        float* mu_yy,
        float* mu_zz,
        float* tempMask,
        float* msat0T0Mask,

        float muMul_xx,
        float muMul_yy,
        float muMul_zz,

        float KB2tempMul_mu0VgammaDtMSatMul,
        hipStream_t stream,
        int Npart)
{

    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    temperature_scaleAnisKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (
        hx, hy, hz,
        mu_xx,
        mu_yy,
        mu_zz,
        tempMask,
        msat0T0Mask,
        muMul_xx,
        muMul_yy,
        muMul_zz,
        KB2tempMul_mu0VgammaDtMSatMul,
        Npart);
}

#ifdef __cplusplus
}
#endif
