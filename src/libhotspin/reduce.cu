#include "hip/hip_runtime.h"
// The code in this source file is based on the reduction code from the CUDPP library. Hence the following notice:

/*
Copyright (c) 2007-2010 The Regents of the University of California, Davis
campus ("The Regents") and NVIDIA Corporation ("NVIDIA"). All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright notice,
      this list of conditions and the following disclaimer in the documentation
      and/or other materials provided with the distribution.
    * Neither the name of the The Regents, nor NVIDIA, nor the names of its
      contributors may be used to endorse or promote products derived from this
      software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// This code has been significantly modified from its original version by Arne Vansteenkiste, 2011.
//  - restricted to use only doubles
//  - more reduction operations than the original "sum" have been added (min, max, maxabs, ...)
//  - added streams for asynchronous execution
// Note that you have to comply with both the above BSD and GPL licences.

//  This file is part of MuMax, a high-performance micromagnetic simulator.
//  Copyright 2011  Arne Vansteenkiste and Ben Van de Wiele.
//  Use of this source code is governed by the GNU General Public License version 3
//  (as published by the Free Software Foundation) that can be found in the license.txt file.
//  Note that you are welcome to modify this code under the condition that you do not remove any
//  copyright notices and prominently state that you modified it, giving a relevant date.

///@todo case 1024 should be added to take advantage of modern GPUs

#include "reduce.h"
#include "gpu_safe.h"

extern "C"
bool isPow2(unsigned int x)
{
    return ((x & (x - 1)) == 0);
}

/// @internal
/// Utility class used to avoid linker errors with extern
/// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

//________________________________________________________________________________________________________________ kernels


/// This kernel takes a partial sum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_sum_kernel(double* g_idata, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i + blockSize];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            mySum = mySum + sdata[tid + 512];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            mySum = mySum + sdata[tid + 256];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            mySum = mySum + sdata[tid + 128];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            mySum = mySum + sdata[tid +  64];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            mySum = mySum + smem[tid + 32];
            smem[tid] = mySum;
        }
        if (blockSize >=  32)
        {
            mySum = mySum + smem[tid + 16];
            smem[tid] = mySum;
        }
        if (blockSize >=  16)
        {
            mySum = mySum + smem[tid +  8];
            smem[tid] = mySum;
        }
        if (blockSize >=   8)
        {
            mySum = mySum + smem[tid +  4];
            smem[tid] = mySum;
        }
        if (blockSize >=   4)
        {
            mySum = mySum + smem[tid +  2];
            smem[tid] = mySum;
        }
        if (blockSize >=   2)
        {
            mySum = mySum + smem[tid +  1];
            smem[tid] = mySum;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}

/// Single-precision dot product
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_sdot_kernel(double* g_idata, double* g_idata2, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i] * g_idata2[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i + blockSize] * g_idata2[i + blockSize];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            mySum = mySum + sdata[tid + 512];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            mySum = mySum + sdata[tid + 256];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            mySum = mySum + sdata[tid + 128];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            mySum = mySum + sdata[tid +  64];
            sdata[tid] = mySum;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            mySum = mySum + smem[tid + 32];
            smem[tid] = mySum;
        }
        if (blockSize >=  32)
        {
            mySum = mySum + smem[tid + 16];
            smem[tid] = mySum;
        }
        if (blockSize >=  16)
        {
            mySum = mySum + smem[tid +  8];
            smem[tid] = mySum;
        }
        if (blockSize >=   8)
        {
            mySum = mySum + smem[tid +  4];
            smem[tid] = mySum;
        }
        if (blockSize >=   4)
        {
            mySum = mySum + smem[tid +  2];
            smem[tid] = mySum;
        }
        if (blockSize >=   2)
        {
            mySum = mySum + smem[tid +  1];
            smem[tid] = mySum;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial maximum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_max_kernel(double* g_idata, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMax = -6E38;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        myMax = fmax(myMax, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            myMax = fmax(myMax, g_idata[i + blockSize]);
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMax;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMax = fmax(myMax, sdata[tid + 256]);
            sdata[tid] = myMax;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMax = fmax(myMax, sdata[tid + 256]);
            sdata[tid] = myMax;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMax = fmax(myMax, sdata[tid + 128]);
            sdata[tid] = myMax;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMax = fmax(myMax, sdata[tid +  64]);
            sdata[tid] = myMax;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMax = fmax(myMax, smem[tid + 32]);
            smem[tid] = myMax;
        }
        if (blockSize >=  32)
        {
            myMax = fmax(myMax, smem[tid + 16]);
            smem[tid] = myMax;
        }
        if (blockSize >=  16)
        {
            myMax = fmax(myMax, smem[tid +  8]);
            smem[tid] = myMax;
        }
        if (blockSize >=   8)
        {
            myMax = fmax(myMax, smem[tid +  4]);
            smem[tid] = myMax;
        }
        if (blockSize >=   4)
        {
            myMax = fmax(myMax, smem[tid +  2]);
            smem[tid] = myMax;
        }
        if (blockSize >=   2)
        {
            myMax = fmax(myMax, smem[tid +  1]);
            smem[tid] = myMax;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial minimum
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_min_kernel(double* g_idata, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMin = 6E38;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        myMin = fmin(myMin, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            myMin = fmin(myMin, g_idata[i + blockSize]);
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMin;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMin = fmin(myMin, sdata[tid + 256]);
            sdata[tid] = myMin;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMin = fmin(myMin, sdata[tid + 256]);
            sdata[tid] = myMin;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMin = fmin(myMin, sdata[tid + 128]);
            sdata[tid] = myMin;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMin = fmin(myMin, sdata[tid +  64]);
            sdata[tid] = myMin;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMin = fmin(myMin, smem[tid + 32]);
            smem[tid] = myMin;
        }
        if (blockSize >=  32)
        {
            myMin = fmin(myMin, smem[tid + 16]);
            smem[tid] = myMin;
        }
        if (blockSize >=  16)
        {
            myMin = fmin(myMin, smem[tid +  8]);
            smem[tid] = myMin;
        }
        if (blockSize >=   8)
        {
            myMin = fmin(myMin, smem[tid +  4]);
            smem[tid] = myMin;
        }
        if (blockSize >=   4)
        {
            myMin = fmin(myMin, smem[tid +  2]);
            smem[tid] = myMin;
        }
        if (blockSize >=   2)
        {
            myMin = fmin(myMin, smem[tid +  1]);
            smem[tid] = myMin;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial maximum of absolute values
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxabs_kernel(double* g_idata, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMaxabs = 0.;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        myMaxabs = fmax(myMaxabs, fabs(g_idata[i]));
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            myMaxabs = fmax(myMaxabs, fabs(g_idata[i + blockSize]));
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMaxabs;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 128]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid +  64]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 32]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  32)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 16]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  16)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  8]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   8)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  4]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   4)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  2]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   2)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  1]);
            smem[tid] = myMaxabs;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}



/// This kernel takes a partial maximum difference between two arrays
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxdiff_kernel(double* a, double* b, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMaxabs = 0.;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        myMaxabs = fmax(myMaxabs, fabs(a[i] - b[i]));
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            myMaxabs = fmax(myMaxabs, fabs(a[i + blockSize] - b[i + blockSize]));
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMaxabs;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 128]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid +  64]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 32]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  32)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 16]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  16)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  8]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   8)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  4]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   4)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  2]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   2)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  1]);
            smem[tid] = myMaxabs;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


/// This kernel calculates a partial maximum sum between two arrays
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxsum_kernel(double* a, double* b, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMaxabs = 0.;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        myMaxabs = fmax(myMaxabs, fabs(a[i] + b[i]));
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            myMaxabs = fmax(myMaxabs, fabs(a[i + blockSize] + b[i + blockSize]));
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMaxabs;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 128]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid +  64]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 32]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  32)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 16]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  16)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  8]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   8)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  4]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   4)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  2]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   2)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  1]);
            smem[tid] = myMaxabs;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}



/// This kernel takes a partial maximum euclidian norm squared of x,y,z component arrays
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxnorm3sq_kernel(double* x, double* y, double* z, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMaxabs = 0.;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        double X = x[i];
        double Y = y[i];
        double Z = z[i];
        myMaxabs = fmax(myMaxabs, (X * X + Y * Y + Z * Z));
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
        {
            X = x[i + blockSize];
            Y = y[i + blockSize];
            Z = z[i + blockSize];
            myMaxabs = fmax(myMaxabs, (X * X + Y * Y + Z * Z));
        }
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMaxabs;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 128]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid +  64]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 32]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  32)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 16]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  16)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  8]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   8)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  4]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   4)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  2]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   2)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  1]);
            smem[tid] = myMaxabs;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


/// This kernel takes a partial maximum euclidian norm squared of the difference between 3-vectors
template <unsigned int blockSize, bool nIsPow2>
__global__ void _gpu_maxnorm3sqdiff_kernel(double* x1, double* y1, double* z1, double* x2, double* y2, double* z2, double* g_odata, unsigned int n)
{
    double* sdata = SharedMemory<double>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMaxabs = 0.;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        double X = x1[i] - x2[i];
        double Y = y1[i] - y2[i];
        double Z = z1[i] - z2[i];
        myMaxabs = fmax(myMaxabs, (X * X + Y * Y + Z * Z));
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
        {
            X = x1[i + blockSize] - x2[i + blockSize];
            Y = y1[i + blockSize] - y2[i + blockSize];
            Z = z1[i + blockSize] - z2[i + blockSize];
            myMaxabs = fmax(myMaxabs, (X * X + Y * Y + Z * Z));
        }
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = myMaxabs;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 1024)
    {
        if (tid < 512)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 512)
    {
        if (tid < 256)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 256]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (tid < 128)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid + 128]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (tid <  64)
        {
            myMaxabs = fmax(myMaxabs, sdata[tid +  64]);
            sdata[tid] = myMaxabs;
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double* smem = sdata;
        if (blockSize >=  64)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 32]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  32)
        {
            myMaxabs = fmax(myMaxabs, smem[tid + 16]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=  16)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  8]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   8)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  4]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   4)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  2]);
            smem[tid] = myMaxabs;
        }
        if (blockSize >=   2)
        {
            myMaxabs = fmax(myMaxabs, smem[tid +  1]);
            smem[tid] = myMaxabs;
        }
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}


//________________________________________________________________________________________________________________ kernel wrappers

#ifdef __cplusplus
extern "C" {
#endif

// single-GPU
__export__ void partialSumAsync1(double* d_idata, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_sum_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_sum_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_sum_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_sum_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_sum_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_sum_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_sum_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_sum_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_sum_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_sum_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_sum_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_sum_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_sum_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_sum_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_sum_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_sum_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_sum_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_sum_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_sum_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_sum_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_sum_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_sum_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
}

__export__ void partialSumAsync(double* input, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialSumAsync1(input, output, blocks, threadsPerBlock, N, stream);
}


// single-GPU
void partialSDotAsync1(double* d_idata, double* d_idata2, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_sdot_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 512:
            _gpu_sdot_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 256:
            _gpu_sdot_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 128:
            _gpu_sdot_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  64:
            _gpu_sdot_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  32:
            _gpu_sdot_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  16:
            _gpu_sdot_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   8:
            _gpu_sdot_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   4:
            _gpu_sdot_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   2:
            _gpu_sdot_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   1:
            _gpu_sdot_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_sdot_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 512:
            _gpu_sdot_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 256:
            _gpu_sdot_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case 128:
            _gpu_sdot_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  64:
            _gpu_sdot_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  32:
            _gpu_sdot_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case  16:
            _gpu_sdot_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   8:
            _gpu_sdot_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   4:
            _gpu_sdot_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   2:
            _gpu_sdot_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        case   1:
            _gpu_sdot_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_idata2, d_odata, size);
            break;
        }
    }
}

void partialSDotAsync(double* input1, double* input2, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialSDotAsync1(input1, input2, output, blocks, threadsPerBlock, N, stream);
}


// single-GPU
__export__ void partialMaxAsync1(double* d_idata, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_max_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_max_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_max_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_max_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_max_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_max_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_max_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_max_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_max_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_max_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_max_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_max_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_max_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_max_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_max_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_max_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_max_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_max_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_max_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_max_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_max_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_max_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
}


__export__ void partialMaxAsync(double* input, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialMaxAsync1(input, output, blocks, threadsPerBlock, N, stream);
}





__export__ void partialMinAsync1(double* d_idata, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_min_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_min_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_min_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_min_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_min_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_min_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_min_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_min_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_min_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_min_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_min_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_min_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_min_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_min_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_min_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_min_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_min_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_min_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_min_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_min_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_min_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_min_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
}


__export__ void partialMinAsync(double* input, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialMinAsync1(input, output, blocks, threadsPerBlock, N, stream);
}




// Single-GPU
__export__ void partialMaxAbsAsync1(double* d_idata, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxabs_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_maxabs_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_maxabs_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_maxabs_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_maxabs_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_maxabs_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_maxabs_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_maxabs_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_maxabs_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_maxabs_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_maxabs_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxabs_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 512:
            _gpu_maxabs_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 256:
            _gpu_maxabs_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case 128:
            _gpu_maxabs_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  64:
            _gpu_maxabs_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  32:
            _gpu_maxabs_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case  16:
            _gpu_maxabs_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   8:
            _gpu_maxabs_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   4:
            _gpu_maxabs_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   2:
            _gpu_maxabs_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        case   1:
            _gpu_maxabs_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
            break;
        }
    }
}


__export__ void partialMaxAbsAsync(double* input, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialMaxAbsAsync1(input, output, blocks, threadsPerBlock, N, stream);
}



// Single-GPU
__export__ void partialMaxDiffAsync1(double* a, double* b, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxdiff_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 512:
            _gpu_maxdiff_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 256:
            _gpu_maxdiff_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 128:
            _gpu_maxdiff_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  64:
            _gpu_maxdiff_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  32:
            _gpu_maxdiff_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  16:
            _gpu_maxdiff_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   8:
            _gpu_maxdiff_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   4:
            _gpu_maxdiff_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   2:
            _gpu_maxdiff_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   1:
            _gpu_maxdiff_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxdiff_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 512:
            _gpu_maxdiff_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 256:
            _gpu_maxdiff_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 128:
            _gpu_maxdiff_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  64:
            _gpu_maxdiff_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  32:
            _gpu_maxdiff_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  16:
            _gpu_maxdiff_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   8:
            _gpu_maxdiff_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   4:
            _gpu_maxdiff_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   2:
            _gpu_maxdiff_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   1:
            _gpu_maxdiff_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        }
    }
}


__export__ void partialMaxDiffAsync(double* a, double* b, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialMaxDiffAsync1(a, b, output, blocks, threadsPerBlock, N, stream);
}

// Single-GPU
__export__ void partialMaxSumAsync1(double* a, double* b, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxsum_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 512:
            _gpu_maxsum_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 256:
            _gpu_maxsum_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 128:
            _gpu_maxsum_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  64:
            _gpu_maxsum_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  32:
            _gpu_maxsum_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  16:
            _gpu_maxsum_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   8:
            _gpu_maxsum_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   4:
            _gpu_maxsum_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   2:
            _gpu_maxsum_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   1:
            _gpu_maxsum_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxsum_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 512:
            _gpu_maxsum_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 256:
            _gpu_maxsum_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case 128:
            _gpu_maxsum_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  64:
            _gpu_maxsum_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  32:
            _gpu_maxsum_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case  16:
            _gpu_maxsum_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   8:
            _gpu_maxsum_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   4:
            _gpu_maxsum_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   2:
            _gpu_maxsum_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        case   1:
            _gpu_maxsum_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(a, b, d_odata, size);
            break;
        }
    }
}


__export__ void partialMaxSumAsync(double* a, double* b, double* output, int blocks, int threadsPerBlock, int N, hipStream_t stream)
{
    partialMaxSumAsync1(a, b, output, blocks, threadsPerBlock, N, stream);
}


// Single-GPU
__export__ void partialMaxNorm3SqAsync1(double* x, double* y, double* z, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxnorm3sq_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 512:
            _gpu_maxnorm3sq_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 256:
            _gpu_maxnorm3sq_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 128:
            _gpu_maxnorm3sq_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  64:
            _gpu_maxnorm3sq_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  32:
            _gpu_maxnorm3sq_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  16:
            _gpu_maxnorm3sq_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   8:
            _gpu_maxnorm3sq_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   4:
            _gpu_maxnorm3sq_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   2:
            _gpu_maxnorm3sq_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   1:
            _gpu_maxnorm3sq_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxnorm3sq_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 512:
            _gpu_maxnorm3sq_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 256:
            _gpu_maxnorm3sq_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case 128:
            _gpu_maxnorm3sq_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  64:
            _gpu_maxnorm3sq_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  32:
            _gpu_maxnorm3sq_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case  16:
            _gpu_maxnorm3sq_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   8:
            _gpu_maxnorm3sq_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   4:
            _gpu_maxnorm3sq_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   2:
            _gpu_maxnorm3sq_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        case   1:
            _gpu_maxnorm3sq_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x, y, z, d_odata, size);
            break;
        }
    }
}


__export__ void partialMaxNorm3SqAsync(double* x, double* y, double* z, double* output, int blocksPerGPU, int threadsPerBlockPerGPU, int NPerGPU, hipStream_t streams)
{
    partialMaxNorm3SqAsync1(x, y, z, output, blocksPerGPU, threadsPerBlockPerGPU, NPerGPU, streams);
}


// Single-GPU
__export__ void partialMaxNorm3SqDiffAsync1(double* x1, double* y1, double* z1, double* x2, double* y2, double* z2, double* d_odata, int blocks, int threads, int size, hipStream_t stream)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

    if (isPow2(size))
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxnorm3sqdiff_kernel<1024, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 512:
            _gpu_maxnorm3sqdiff_kernel<512, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 256:
            _gpu_maxnorm3sqdiff_kernel<256, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 128:
            _gpu_maxnorm3sqdiff_kernel<128, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  64:
            _gpu_maxnorm3sqdiff_kernel< 64, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  32:
            _gpu_maxnorm3sqdiff_kernel< 32, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  16:
            _gpu_maxnorm3sqdiff_kernel< 16, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   8:
            _gpu_maxnorm3sqdiff_kernel<  8, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   4:
            _gpu_maxnorm3sqdiff_kernel<  4, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   2:
            _gpu_maxnorm3sqdiff_kernel<  2, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   1:
            _gpu_maxnorm3sqdiff_kernel<  1, true> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        }
    }
    else
    {
        switch (threads)
        {
        case 1024:
            _gpu_maxnorm3sqdiff_kernel<1024, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 512:
            _gpu_maxnorm3sqdiff_kernel<512, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 256:
            _gpu_maxnorm3sqdiff_kernel<256, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case 128:
            _gpu_maxnorm3sqdiff_kernel<128, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  64:
            _gpu_maxnorm3sqdiff_kernel< 64, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  32:
            _gpu_maxnorm3sqdiff_kernel< 32, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case  16:
            _gpu_maxnorm3sqdiff_kernel< 16, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   8:
            _gpu_maxnorm3sqdiff_kernel<  8, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   4:
            _gpu_maxnorm3sqdiff_kernel<  4, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   2:
            _gpu_maxnorm3sqdiff_kernel<  2, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        case   1:
            _gpu_maxnorm3sqdiff_kernel<  1, false> <<< dimGrid, dimBlock, smemSize, stream>>>(x1, y1, z1, x2, y2, z2, d_odata, size);
            break;
        }
    }
}

__export__ void partialMaxNorm3SqDiffAsync(double* x1, double* y1, double* z1, double* x2, double* y2, double* z2, double* output, int blocksPerGPU, int threadsPerBlockPerGPU, int NPerGPU, hipStream_t streams)
{	
    partialMaxNorm3SqDiffAsync1(x1, y1, z1, x2, y2, z2, output, blocksPerGPU, threadsPerBlockPerGPU, NPerGPU, streams);
}


#ifdef __cplusplus
}
#endif
