#include "hip/hip_runtime.h"
#include "Qinter.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__global__ void QinterKern(float* __restrict__ Qi,
                           const float* __restrict__ Ti, const float* __restrict__ Tj,
                           const float* __restrict__ GijMsk,
                           const float GijMul,
                           int Npart)
{

    int i = threadindex;
    if (i < Npart)
    {
        float Tii = Ti[i];
        float Tjj = Tj[i];
        float Gij = (GijMsk == NULL) ? GijMul : GijMul * GijMsk[i];
        Qi[i] = Gij * (Tjj - Tii);
    }
}

__export__ void QinterAsync(float* Qi,
                            float* Ti, float* Tj,
                            float* Gij,
                            float GijMul,
                            int Npart,
                            hipStream_t stream)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    QinterKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (Qi,
            Ti,
            Tj,
            Gij,
            GijMul,
            Npart);
}

#ifdef __cplusplus
}
#endif

