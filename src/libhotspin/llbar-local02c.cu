#include "hip/hip_runtime.h"
#include "llbar-local02c.h"
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "common_func.h"
#ifdef __cplusplus
extern "C" {
#endif

__global__ void llbarLocal02cKern(double* __restrict__ tx, double* __restrict__ ty, double* __restrict__ tz,
                                  double* __restrict__ mx, double* __restrict__ my, double* __restrict__ mz,
                                  double* __restrict__ hx, double* __restrict__ hy, double* __restrict__ hz,

                                  double* __restrict__ msat0T0Msk,

                                  double* __restrict__ mu_xx,
                                  double* __restrict__ mu_yy,
                                  double* __restrict__ mu_zz,

                                  const double muMul_xx,
                                  const double muMul_yy,
                                  const double muMul_zz,

                                  int Npart)
{

    int x0 = threadindex;

    if (x0 < Npart)
    {

        double msat0T0 = (msat0T0Msk == NULL) ? 1.0 : msat0T0Msk[x0];
        double3 m = make_double3(mx[x0], my[x0], mz[x0]);

        // make sure there is no torque for non-magnetic points
        if (msat0T0 == 0.0)
        {
            tx[x0] = 0.0;
            ty[x0] = 0.0;
            tz[x0] = 0.0;
            return;
        }

        double3 H = make_double3(hx[x0], hy[x0], hz[x0]);

        double3 mxH = cross(m, H);

        double3 mu_mxH;

        double m_xx = (mu_xx != NULL) ? mu_xx[x0] * muMul_xx : muMul_xx;

        mu_mxH.x = m_xx * mxH.x;

        double m_yy = (mu_yy != NULL) ? mu_yy[x0] * muMul_yy : muMul_yy;

        mu_mxH.y = m_yy * mxH.y;

        double m_zz = (mu_zz != NULL) ? mu_zz[x0] * muMul_zz : muMul_zz;

        mu_mxH.z = m_zz * mxH.z;

        double3 _mxmu_mxH = cross(mu_mxH, m);

        tx[x0] = _mxmu_mxH.x;
        ty[x0] = _mxmu_mxH.y;
        tz[x0] = _mxmu_mxH.z;
    }
}

__export__  void llbar_local02c_async(double* tx, double*  ty, double*  tz,
                                      double*  mx, double*  my, double*  mz,
                                      double*  hx, double*  hy, double*  hz,

                                      double* msat0T0,

                                      double* mu_xx,
                                      double* mu_yy,
                                      double* mu_zz,

                                      const double muMul_xx,
                                      const double muMul_yy,
                                      const double muMul_zz,

                                      hipStream_t stream,
                                      int Npart)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);

    llbarLocal02cKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (tx, ty, tz,
            mx, my, mz,
            hx, hy, hz,

            msat0T0,

            mu_xx,
            mu_yy,
            mu_zz,

            muMul_xx,
            muMul_yy,
            muMul_zz,

            Npart);

}

// ========================================

#ifdef __cplusplus
}
#endif
