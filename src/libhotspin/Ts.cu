#include "hip/hip_runtime.h"
#include "brillouin.h"

#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__device__ double findroot_Ridders_Ts(funcTs* f, double J, double mult, double C, double xa, double xb)
{

    double ya = f[0](xa, J, mult, C);
    if (fabs(ya) < zero) return xa;
    double yb = f[0](xb, J, mult, C);
    if (fabs(yb) < zero) return xb;

    double y1 = ya;
    double x1 = xa;
    double y2 = yb;
    double x2 = xb;

    double x = 1.0e10;
    double y = 1.0e10;
    double tx = x;

    double teps = x;

    double x3 = 0.0;
    double y3 = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    int iter = 0;
    while (teps > eps && iter < 1000)
    {

        x3 = 0.5 * (x2 + x1);
        y3 = f[0](x3, J, mult, C);

        dy = (y3 * y3 - y1 * y2);
        if (dy == 0.0)
        {
            x = x3;
            break;
        }

        dx = (x3 - x1) * sign(y1 - y2) * y3 / (sqrt(dy));

        x = x3 + dx;
        y = f[0](x, J, mult, C);

        y2 = (signbit(y) == signbit(y3)) ? y2 : y3;
        x2 = (signbit(y) == signbit(y3)) ? x2 : x3;

        y2 = (signbit(y) == signbit(y1) || x2 == x3) ? y2 : y1;
        x2 = (signbit(y) == signbit(y1) || x2 == x3) ? x2 : x1;

        y1 = y;
        x1 = x;

        teps = fabs((x - tx) / (tx + x));

        tx = x;
        iter++;

    }
    return x;
}

// here n = <Sz>/ S
// <Sz> = n * S
// <Sz> = S * Bj(S*J0*<Sz>/(kT))

__device__ double ModelTs(double n, double J, double pre, double C)
{
    double x = (n == 0.0) ? 1.0e38 : pre / n;
    double val = Bj(J, x) - C;
    return val;
}

__device__ funcTs pModelTs = ModelTs;

__global__ void tsKern(double* __restrict__ Ts,
                      double* __restrict__ msatMsk,
                      double* __restrict__ msat0T0Msk,
                      double* __restrict__ TcMsk,
                      double* __restrict__ SMsk,
                      const double msatMul,
                      const double msat0T0Mul,
                      const double TcMul,
                      const double SMul,
                      int Npart)
{
    int i = threadindex;
    if (i < Npart)
    {

        double msat0T0 = msat0T0Mul * getMaskUnity(msat0T0Msk, i);
        if (msat0T0 == 0.0)
        {
            Ts[i] = 0.0;
            return;
        }

        double msat = msatMul * getMaskUnity(msatMsk, i);
        if (msat == msat0T0) {
        	Ts[i] = 0.0;
        	return;
        }

        double Tc = TcMul * getMaskUnity(TcMsk, i);
        if (msat == 0.0)
        {
            Ts[i] = Tc;
            return;
        }

        double S  = (SMsk  == NULL) ? SMul  : SMul  * SMsk[i];

        double J0  = 3.0 * Tc / (S * (S + 1.0));
        double m = msat / msat0T0;
        double pre = S * S * J0 * m;
        double T = findroot_Ridders_Ts(&pModelTs, S, pre, m, 0.0, Tc);

        Ts[i] = (double)T;
    }
}

__export__ void tsAsync(double* Ts,
                              double* msat,
                              double* msat0T0,
                              double* Tc,
                              double* S,
                              const double msatMul,
                              const double msat0T0Mul,
                              const double TcMul,
                              const double SMul,
                              int Npart,
                              hipStream_t stream)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    tsKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (Ts,
            msat,
            msat0T0,
            Tc,
            S,
            msatMul,
            msat0T0Mul,
            TcMul,
            SMul,
            Npart);
}

#ifdef __cplusplus
}
#endif
