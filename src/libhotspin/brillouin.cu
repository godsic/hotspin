#include "hip/hip_runtime.h"
#include "brillouin.h"

#include <hip/hip_runtime.h>
#include "gpu_conf.h"
#include "gpu_safe.h"
#include "common_func.h"

#ifdef __cplusplus
extern "C" {
#endif

///@internal
__device__ double findroot_Ridders(func* f, double J, double mult, double xa, double xb)
{

    double ya = f[0](xa, J, mult);
    if (fabs(ya) < zero) return xa;
    double yb = f[0](xb, J, mult);
    if (fabs(yb) < zero) return xb;

    double y1 = ya;
    double x1 = xa;
    double y2 = yb;
    double x2 = xb;

    double x = 1.0e10;
    double y = 1.0e10;
    double tx = x;

    double teps = x;

    double x3 = 0.0;
    double y3 = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    int iter = 0;
    while (teps > eps && iter < 1000)
    {

        x3 = 0.5 * (x2 + x1);
        y3 = f[0](x3, J, mult);

        dy = (y3 * y3 - y1 * y2);
        if (dy == 0.0)
        {
            x = x3;
            break;
        }

        dx = (x3 - x1) * sign(y1 - y2) * y3 / (sqrt(dy));

        x = x3 + dx;
        y = f[0](x, J, mult);

        y2 = (signbit(y) == signbit(y3)) ? y2 : y3;
        x2 = (signbit(y) == signbit(y3)) ? x2 : x3;

        y2 = (signbit(y) == signbit(y1) || x2 == x3) ? y2 : y1;
        x2 = (signbit(y) == signbit(y1) || x2 == x3) ? x2 : x1;

        y1 = y;
        x1 = x;

        teps = fabs((x - tx) / (tx + x));

        tx = x;
        iter++;

    }
    return x;
}


// here n = m / me
// <Sz> = n * J
// <Sz> = J * Bj(S*J0*<Sz>/(kT))

__device__ double Model(double n, double J, double pre)
{
    double x = pre * n;
    double val = (J > INFINITESPINLIMIT) ? L(x) : Bj(J, x);
    val = val - n;
    return val;
}

__device__ func pModel = Model;

__global__ void brillouinKern(double* __restrict__ msat0Msk,
                              double* __restrict__ msat0T0Msk,
                              double* __restrict__ T,
                              double* __restrict__ TcMsk,
                              double* __restrict__ SMsk,
                              const double msat0Mul,
                              const double msat0T0Mul,
                              const double TcMul,
                              const double SMul,
                              int Npart)
{
    int i = threadindex;
    if (i < Npart)
    {
        double Temp = T[i];

        double msat0T0 = msat0T0Mul * getMaskUnity(msat0T0Msk, i);

        if (msat0T0 == 0.0)
        {
            msat0Msk[i] = 0.0;
            return;
        }

        if (Temp == 0.0)
        {
            msat0Msk[i] = msat0T0 / msat0Mul;
            return;
        }

        double Tc = TcMul * getMaskUnity(TcMsk, i);

        if (Temp > Tc)
        {
            msat0Msk[i] = 0.0;
            return;
        }

        double S  = SMul * getMaskUnity(SMsk, i);

        double preS = (S > INFINITESPINLIMIT) ? 1.0 : S / (S + 1.0);
        double pre = 3.0 * preS * (Tc / Temp);

        double dT = (Tc - Temp) / Tc;
        double lowLimit = (dT < 0.0004) ? -0.1 : 0.01;
        double hiLimit  = (dT < 0.0004) ?  0.5 : 1.1;
        double msat0 = findroot_Ridders(&pModel, S, pre, lowLimit, hiLimit);

        msat0Msk[i] = msat0T0 * fabs(msat0) / (msat0Mul);
    }
}

__export__ void brillouinAsync(double* msat0,
                               double* msat0T0,
                               double* T,
                               double* Tc,
                               double* S,
                               const double msat0Mul,
                               const double msat0T0Mul,
                               const double TcMul,
                               const double SMul,
                               int Npart,
                               hipStream_t stream)
{
    dim3 gridSize, blockSize;
    make1dconf(Npart, &gridSize, &blockSize);
    brillouinKern <<< gridSize, blockSize, 0, hipStream_t(stream)>>> (msat0,
            msat0T0,
            T,
            Tc,
            S,
            msat0Mul,
            msat0T0Mul,
            TcMul,
            SMul,
            Npart);
}

#ifdef __cplusplus
}
#endif
